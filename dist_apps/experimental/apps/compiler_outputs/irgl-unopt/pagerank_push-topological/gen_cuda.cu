#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=1 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=False $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_NOUT;
float * P_RESIDUAL;
float * P_VALUE;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
__global__ void ResetGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, unsigned int * p_nout, float * p_residual, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_value[src] = 0;
      p_nout[src] = 0;
      p_residual[src] = 0;
    }
  }
  // FP: "9 -> 10;
}
__global__ void InitializeGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float  local_alpha, unsigned int * p_nout, float * p_residual, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  float delta;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    index_type nbr_end;
    bool pop  = src < __end;
    if (pop)
    {
      p_value[src] = local_alpha;
      p_nout[src] = graph.getOutDegree(src);
      if (p_nout[src] > 0)
      {
        delta = p_value[src]*(1-local_alpha)/p_nout[src];
      }
      else
      {
        pop = false;
      }
    }
    if (!pop)
    {
      continue;
    }
    nbr_end = (graph).getFirstEdge((src) + 1);
    for (index_type nbr = (graph).getFirstEdge(src) + 0; nbr < nbr_end; nbr += 1)
    {
      index_type dst;
      dst = graph.getAbsDestination(nbr);
      atomicAdd(&p_residual[dst], delta);
    }
  }
  // FP: "21 -> 22;
}
__global__ void PageRank(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float  local_alpha, float local_tolerance, unsigned int * p_nout, float * p_residual, float * p_value, Sum ret_val)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  typedef hipcub::BlockReduce<int, TB_SIZE> _br;
  __shared__ _br::TempStorage _ts;
  ret_val.thread_entry();
  float residual_old;
  float delta;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    index_type nbr_end;
    bool pop  = src < __end;
    if (pop)
    {
      residual_old = atomicExch(&p_residual[src], 0.0);
      p_value[src] += residual_old;
      if (p_nout[src] > 0)
      {
        delta = residual_old*(1-local_alpha)/p_nout[src];
      }
      else
      {
        pop = false;
      }
    }
    if (!pop)
    {
      continue;
    }
    nbr_end = (graph).getFirstEdge((src) + 1);
    for (index_type nbr = (graph).getFirstEdge(src) + 0; nbr < nbr_end; nbr += 1)
    {
      index_type dst;
      float dst_residual_old;
      dst = graph.getAbsDestination(nbr);
      dst_residual_old = atomicAdd(&p_residual[dst], delta);
      if ((dst_residual_old <= local_tolerance) && ((dst_residual_old + delta) >= local_tolerance))
      {
        ret_val.do_return( 1);
        continue;
      }
    }
  }
  ret_val.thread_exit<_br>(_ts);
}
void ResetGraph_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  ResetGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, ctx->nout.gpu_wr_ptr(), ctx->residual.gpu_wr_ptr(), ctx->value.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void ResetGraph_all_cuda(struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->nowned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, const float & local_alpha, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, local_alpha, ctx->nout.gpu_wr_ptr(), ctx->residual.gpu_wr_ptr(), ctx->value.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_all_cuda(const float & local_alpha, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->nowned, local_alpha, ctx);
  // FP: "2 -> 3;
}
void PageRank_cuda(unsigned int  __begin, unsigned int  __end, int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<int> retval = Shared<int>(1);
  Sum _rv;
  *(retval.cpu_wr_ptr()) = 0;
  _rv.rv = retval.gpu_wr_ptr();
  PageRank <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, local_alpha, local_tolerance, ctx->nout.gpu_wr_ptr(), ctx->residual.gpu_wr_ptr(), ctx->value.gpu_wr_ptr(), _rv);
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
  __retval = *(retval.cpu_rd_ptr());
  // FP: "7 -> 8;
}
void PageRank_all_cuda(int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->nowned, __retval, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}