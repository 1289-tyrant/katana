#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=True $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_DIST_CURRENT;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
static const int __tb_BFS = TB_SIZE;
__global__ void InitializeGraph(CSRGraph graph, int  nowned, unsigned int local_infinity, unsigned int local_src_node, unsigned int * p_dist_current)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    p_dist_current[src] = (graph.node_data[src] == local_src_node) ? 0 : local_infinity;
  }
}
__global__ void BFS(CSRGraph graph, int  nowned, unsigned int * p_dist_current, Worklist2 in_wl, Worklist2 out_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_BFS;
  if (tid == 0)
    in_wl.reset_next_slot();

  index_type wlvertex_end;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  __shared__ npsTy nps ;
  wlvertex_end = roundup((*((volatile index_type *) (in_wl).dindex)), (blockDim.x));
  for (index_type wlvertex = 0 + tid; wlvertex < wlvertex_end; wlvertex += nthreads)
  {
    int src;
    bool pop;
    unsigned int sdist;
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    pop = (in_wl).pop_id(wlvertex, src);
    sdist = p_dist_current[src];
    struct NPInspector1 _np = {0,0,0,0,0,0};
    __shared__ struct { unsigned int sdist; } _np_closure [TB_SIZE];
    _np_closure[threadIdx.x].sdist = sdist;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    __syncthreads();
    while (true)
    {
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      __syncthreads();
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        __syncthreads();
        break;
      }
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      __syncthreads();
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      assert(nps.tb.src < __kernel_tb_size);
      sdist = _np_closure[nps.tb.src].sdist;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type jj;
        jj = ns +_np_j;
        {
          index_type dst;
          unsigned int new_dist;
          unsigned int old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = 1 + sdist;
          old_dist = atomicMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            index_type _start_30;
            _start_30 = (out_wl).setup_push_warp_one();;
            (out_wl).do_push(_start_30, 0, dst);
          }
        }
      }
      __syncthreads();
    }

    {
      const int warpid = threadIdx.x / 32;
      const int _np_laneid = hipcub::LaneId();
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        sdist = _np_closure[nps.warp.src[warpid]].sdist;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type jj;
          jj = _np_w_start +_np_ii;
          {
            index_type dst;
            unsigned int new_dist;
            unsigned int old_dist;
            dst = graph.getAbsDestination(jj);
            new_dist = 1 + sdist;
            old_dist = atomicMin(&p_dist_current[dst], new_dist);
            if (old_dist > new_dist)
            {
              index_type _start_30;
              _start_30 = (out_wl).setup_push_warp_one();;
              (out_wl).do_push(_start_30, 0, dst);
            }
          }
        }
      }
      __syncthreads();
    }

    __syncthreads();
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    while (_np.work())
    {
      int _np_i =0;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      __syncthreads();

      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type jj;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        sdist = _np_closure[nps.fg.src[_np_i]].sdist;
        jj= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          unsigned int new_dist;
          unsigned int old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = 1 + sdist;
          old_dist = atomicMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            index_type _start_30;
            _start_30 = (out_wl).setup_push_warp_one();;
            (out_wl).do_push(_start_30, 0, dst);
          }
        }
      }
      _np.execute_round_done(ITSIZE);
      __syncthreads();
    }
    assert(threadIdx.x < __kernel_tb_size);
    sdist = _np_closure[threadIdx.x].sdist;
  }
}
void InitializeGraph_cuda(unsigned int local_src_node, unsigned int local_infinity, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_infinity, local_src_node, ctx->dist_current.gpu_wr_ptr());
  check_cuda_kernel;
}
void BFS_cuda(struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  ctx->in_wl.update_gpu(ctx->shared_wl->num_in_items);
  ctx->out_wl.will_write();
  ctx->out_wl.reset();
  BFS <<<blocks, __tb_BFS>>>(ctx->gg, ctx->nowned, ctx->dist_current.gpu_wr_ptr(), ctx->in_wl, ctx->out_wl);
  check_cuda_kernel;
  ctx->out_wl.update_cpu();
  ctx->shared_wl->num_out_items = ctx->out_wl.nitems();
}