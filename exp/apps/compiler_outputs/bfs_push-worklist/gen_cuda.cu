#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=True $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_DIST_CURRENT;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
static const int __tb_BFS = TB_SIZE;
__global__ void InitializeGraph(CSRGraph graph, int  nowned, const unsigned int  local_infinity, unsigned int local_src_node, unsigned int * p_dist_current)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    p_dist_current[src] = (graph.node_data[src] == local_src_node) ? 0 : local_infinity;
  }
  // FP: "4 -> 5;
}
__global__ void BFS(CSRGraph graph, int  nowned, unsigned int * p_dist_current, Worklist2 in_wl, Worklist2 out_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_BFS;
  if (tid == 0)
    in_wl.reset_next_slot();

  index_type wlvertex_end;
  index_type wlvertex_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  wlvertex_end = *((volatile index_type *) (in_wl).dindex);
  wlvertex_rup = (roundup((*((volatile index_type *) (in_wl).dindex)), (blockDim.x)));
  for (index_type wlvertex = 0 + tid; wlvertex < wlvertex_rup; wlvertex += nthreads)
  {
    int src;
    bool pop;
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    // FP: "7 -> 8;
    // FP: "8 -> 9;
    pop = (in_wl).pop_id(wlvertex, src);
    // FP: "9 -> 10;
    // FP: "11 -> 12;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "12 -> 13;
    __shared__ struct { int src; } _np_closure [TB_SIZE];
    // FP: "13 -> 14;
    _np_closure[threadIdx.x].src = src;
    // FP: "14 -> 15;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "17 -> 18;
    // FP: "18 -> 19;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "19 -> 20;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "20 -> 21;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "23 -> 24;
    __syncthreads();
    // FP: "24 -> 25;
    while (true)
    {
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      __syncthreads();
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        __syncthreads();
        break;
      }
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      __syncthreads();
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type jj;
        jj = ns +_np_j;
        {
          index_type dst;
          unsigned int new_dist;
          unsigned int old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = 1 + p_dist_current[src];
          old_dist = atomicMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            index_type _start_31;
            _start_31 = (out_wl).setup_push_warp_one();;
            (out_wl).do_push(_start_31, 0, dst);
          }
        }
      }
      __syncthreads();
      // FP: "58 -> 25;
    }
    // FP: "59 -> 60;

    // FP: "60 -> 61;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "61 -> 62;
      const int _np_laneid = hipcub::LaneId();
      // FP: "62 -> 63;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type jj;
          jj = _np_w_start +_np_ii;
          {
            index_type dst;
            unsigned int new_dist;
            unsigned int old_dist;
            dst = graph.getAbsDestination(jj);
            new_dist = 1 + p_dist_current[src];
            old_dist = atomicMin(&p_dist_current[dst], new_dist);
            if (old_dist > new_dist)
            {
              index_type _start_31;
              _start_31 = (out_wl).setup_push_warp_one();;
              (out_wl).do_push(_start_31, 0, dst);
            }
          }
        }
      }
      // FP: "87 -> 88;
      __syncthreads();
      // FP: "88 -> 89;
    }

    // FP: "89 -> 90;
    __syncthreads();
    // FP: "90 -> 91;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "91 -> 92;
    while (_np.work())
    {
      // FP: "92 -> 93;
      int _np_i =0;
      // FP: "93 -> 94;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "94 -> 95;
      __syncthreads();
      // FP: "95 -> 96;

      // FP: "96 -> 97;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type jj;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        jj= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          unsigned int new_dist;
          unsigned int old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = 1 + p_dist_current[src];
          old_dist = atomicMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            index_type _start_31;
            _start_31 = (out_wl).setup_push_warp_one();;
            (out_wl).do_push(_start_31, 0, dst);
          }
        }
      }
      // FP: "112 -> 113;
      _np.execute_round_done(ITSIZE);
      // FP: "113 -> 114;
      __syncthreads();
      // FP: "114 -> 92;
    }
    // FP: "115 -> 116;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
    // FP: "116 -> 6;
  }
  // FP: "117 -> 118;
}
void InitializeGraph_cuda(const unsigned int & local_infinity, unsigned int local_src_node, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_infinity, local_src_node, ctx->dist_current.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void BFS_cuda(struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  ctx->in_wl.update_gpu(ctx->shared_wl->num_in_items);
  // FP: "5 -> 6;
  ctx->out_wl.will_write();
  // FP: "6 -> 7;
  ctx->out_wl.reset();
  // FP: "7 -> 8;
  BFS <<<blocks, __tb_BFS>>>(ctx->gg, ctx->nowned, ctx->dist_current.gpu_wr_ptr(), ctx->in_wl, ctx->out_wl);
  // FP: "8 -> 9;
  check_cuda_kernel;
  // FP: "9 -> 10;
  ctx->out_wl.update_cpu();
  // FP: "10 -> 11;
  ctx->shared_wl->num_out_items = ctx->out_wl.nitems();
  // FP: "11 -> 12;
}
