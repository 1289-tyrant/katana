#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
int * P_NOUT;
float * P_SUM;
float * P_VALUE;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
static const int __tb_PageRank_partial = TB_SIZE;
static const int __tb_InitializeGraph = TB_SIZE;
__global__ void ResetGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, int * p_nout, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_value[src] = 0;
      p_nout[src] = 0;
    }
  }
  // FP: "8 -> 9;
}
__global__ void InitializeGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float  local_alpha, int * p_nout, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_InitializeGraph;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
      p_value[src] = local_alpha;
    }
    // FP: "10 -> 11;
    // FP: "13 -> 14;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "14 -> 15;
    __shared__ struct { ; } _np_closure [TB_SIZE];
    // FP: "15 -> 16;
    // FP: "16 -> 17;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "19 -> 20;
    // FP: "20 -> 21;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "21 -> 22;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "22 -> 23;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "25 -> 26;
    __syncthreads();
    // FP: "26 -> 27;
    while (true)
    {
      // FP: "27 -> 28;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "30 -> 31;
      __syncthreads();
      // FP: "31 -> 32;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "32 -> 33;
        __syncthreads();
        // FP: "33 -> 34;
        break;
      }
      // FP: "35 -> 36;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "38 -> 39;
      __syncthreads();
      // FP: "39 -> 40;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "40 -> 41;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "43 -> 44;
      assert(nps.tb.src < __kernel_tb_size);
      // FP: "44 -> 45;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicAdd(&p_nout[dst], 1);
        }
      }
      // FP: "51 -> 52;
      __syncthreads();
    }
    // FP: "53 -> 54;

    // FP: "54 -> 55;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "55 -> 56;
      const int _np_laneid = hipcub::LaneId();
      // FP: "56 -> 57;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            atomicAdd(&p_nout[dst], 1);
          }
        }
      }
      // FP: "73 -> 74;
      __syncthreads();
      // FP: "74 -> 75;
    }

    // FP: "75 -> 76;
    __syncthreads();
    // FP: "76 -> 77;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "77 -> 78;
    while (_np.work())
    {
      // FP: "78 -> 79;
      int _np_i =0;
      // FP: "79 -> 80;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "80 -> 81;
      __syncthreads();
      // FP: "81 -> 82;

      // FP: "82 -> 83;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicAdd(&p_nout[dst], 1);
        }
      }
      // FP: "90 -> 91;
      _np.execute_round_done(ITSIZE);
      // FP: "91 -> 92;
      __syncthreads();
    }
    // FP: "93 -> 94;
    assert(threadIdx.x < __kernel_tb_size);
  }
  // FP: "95 -> 96;
}
__global__ void PageRank_partial(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, int * p_nout, float * p_sum, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_PageRank_partial;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
      p_sum[src] = 0;
    }
    // FP: "10 -> 11;
    // FP: "13 -> 14;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "14 -> 15;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "15 -> 16;
    _np_closure[threadIdx.x].src = src;
    // FP: "16 -> 17;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "19 -> 20;
    // FP: "20 -> 21;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "21 -> 22;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "22 -> 23;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "25 -> 26;
    __syncthreads();
    // FP: "26 -> 27;
    while (true)
    {
      // FP: "27 -> 28;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "30 -> 31;
      __syncthreads();
      // FP: "31 -> 32;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "32 -> 33;
        __syncthreads();
        // FP: "33 -> 34;
        break;
      }
      // FP: "35 -> 36;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "38 -> 39;
      __syncthreads();
      // FP: "39 -> 40;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "40 -> 41;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "43 -> 44;
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "44 -> 45;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          unsigned int dnout;
          dst = graph.getAbsDestination(nbr);
          dnout = p_nout[dst];
          if (dnout > 0)
          {
            p_sum[src] += p_value[dst]/dnout;
          }
        }
      }
      // FP: "55 -> 56;
      __syncthreads();
    }
    // FP: "57 -> 58;

    // FP: "58 -> 59;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "59 -> 60;
      const int _np_laneid = hipcub::LaneId();
      // FP: "60 -> 61;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            unsigned int dnout;
            dst = graph.getAbsDestination(nbr);
            dnout = p_nout[dst];
            if (dnout > 0)
            {
              p_sum[src] += p_value[dst]/dnout;
            }
          }
        }
      }
      // FP: "81 -> 82;
      __syncthreads();
      // FP: "82 -> 83;
    }

    // FP: "83 -> 84;
    __syncthreads();
    // FP: "84 -> 85;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "85 -> 86;
    while (_np.work())
    {
      // FP: "86 -> 87;
      int _np_i =0;
      // FP: "87 -> 88;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "88 -> 89;
      __syncthreads();
      // FP: "89 -> 90;

      // FP: "90 -> 91;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          unsigned int dnout;
          dst = graph.getAbsDestination(nbr);
          dnout = p_nout[dst];
          if (dnout > 0)
          {
            p_sum[src] += p_value[dst]/dnout;
          }
        }
      }
      // FP: "102 -> 103;
      _np.execute_round_done(ITSIZE);
      // FP: "103 -> 104;
      __syncthreads();
    }
    // FP: "105 -> 106;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "107 -> 108;
}
__global__ void PageRank(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float  local_alpha, float local_tolerance, float * p_sum, float * p_value, Any any_retval)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  float pr_value;
  float diff;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      pr_value = p_sum[src]*(1.0 - local_alpha) + local_alpha;
      diff = fabs(pr_value - p_value[src]);
      if (diff > local_tolerance)
      {
        p_value[src] = pr_value;
        any_retval.return_( 1);
      }
    }
  }
  // FP: "14 -> 15;
}
void ResetGraph_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  ResetGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, ctx->nout.gpu_wr_ptr(), ctx->value.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void ResetGraph_all_cuda(struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->nowned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, const float & local_alpha, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, __tb_InitializeGraph>>>(ctx->gg, ctx->nowned, __begin, __end, local_alpha, ctx->nout.gpu_wr_ptr(), ctx->value.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_all_cuda(const float & local_alpha, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->nowned, local_alpha, ctx);
  // FP: "2 -> 3;
}
void PageRank_partial_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  PageRank_partial <<<blocks, __tb_PageRank_partial>>>(ctx->gg, ctx->nowned, __begin, __end, ctx->nout.gpu_wr_ptr(), ctx->sum.gpu_wr_ptr(), ctx->value.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void PageRank_partial_all_cuda(struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  PageRank_partial_cuda(0, ctx->nowned, ctx);
  // FP: "2 -> 3;
}
void PageRank_cuda(unsigned int  __begin, unsigned int  __end, int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  *(ctx->p_retval.cpu_wr_ptr()) = __retval;
  // FP: "5 -> 6;
  ctx->any_retval.rv = ctx->p_retval.gpu_wr_ptr();
  // FP: "6 -> 7;
  PageRank <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, local_alpha, local_tolerance, ctx->sum.gpu_wr_ptr(), ctx->value.gpu_wr_ptr(), ctx->any_retval);
  // FP: "7 -> 8;
  check_cuda_kernel;
  // FP: "8 -> 9;
  __retval = *(ctx->p_retval.cpu_rd_ptr());
  // FP: "9 -> 10;
}
void PageRank_all_cuda(int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->nowned, __retval, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}