#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=1 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=False $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_DIST_CURRENT;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
__global__ void InitializeGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const unsigned int  local_infinity, unsigned int local_src_node, unsigned int * p_dist_current)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_dist_current[src] = (graph.node_data[src] == local_src_node) ? 0 : local_infinity;
    }
  }
  // FP: "7 -> 8;
}
__global__ void SSSP(CSRGraph graph, unsigned int __nowned, unsigned int * p_dist_current, Worklist2 in_wl, Worklist2 out_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  if (tid == 0)
    in_wl.reset_next_slot();

  index_type wlvertex_end;
  // FP: "1 -> 2;
  wlvertex_end = *((volatile index_type *) (in_wl).dindex);
  for (index_type wlvertex = 0 + tid; wlvertex < wlvertex_end; wlvertex += nthreads)
  {
    int src;
    bool pop;
    index_type jj_end;
    pop = (in_wl).pop_id(wlvertex, src);
    if (pop)
    {
    }
    if (!pop)
    {
      continue;
    }
    jj_end = (graph).getFirstEdge((src) + 1);
    for (index_type jj = (graph).getFirstEdge(src) + 0; jj < jj_end; jj += 1)
    {
      index_type dst;
      unsigned int new_dist;
      unsigned int old_dist;
      dst = graph.getAbsDestination(jj);
      new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
      old_dist = atomicMin(&p_dist_current[dst], new_dist);
      if (old_dist > new_dist)
      {
        index_type _start_39;
        _start_39 = (out_wl).setup_push_warp_one();;
        (out_wl).do_push(_start_39, 0, dst);
      }
    }
  }
  // FP: "24 -> 25;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, const unsigned int & local_infinity, unsigned int local_src_node, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, local_infinity, local_src_node, ctx->dist_current.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_all_cuda(const unsigned int & local_infinity, unsigned int local_src_node, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->nowned, local_infinity, local_src_node, ctx);
  // FP: "2 -> 3;
}
void SSSP_cuda(struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  ctx->in_wl.update_gpu(ctx->shared_wl->num_in_items);
  // FP: "5 -> 6;
  ctx->out_wl.will_write();
  // FP: "6 -> 7;
  ctx->out_wl.reset();
  // FP: "7 -> 8;
  SSSP <<<blocks, threads>>>(ctx->gg, ctx->nowned, ctx->dist_current.gpu_wr_ptr(), ctx->in_wl, ctx->out_wl);
  // FP: "8 -> 9;
  check_cuda_kernel;
  // FP: "9 -> 10;
  ctx->out_wl.update_cpu();
  // FP: "10 -> 11;
  ctx->shared_wl->num_out_items = ctx->out_wl.nitems();
  // FP: "11 -> 12;
}