#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_DIST_CURRENT;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
static const int __tb_SSSP = TB_SIZE;
__global__ void InitializeGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const unsigned int  local_infinity, unsigned int local_src_node, unsigned int * p_dist_current)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_dist_current[src] = (graph.node_data[src] == local_src_node) ? 0 : local_infinity;
    }
  }
  // FP: "7 -> 8;
}
__global__ void SSSP(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, unsigned int * p_dist_current, Sum sum_retval)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_SSSP;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
    }
    // FP: "9 -> 10;
    // FP: "12 -> 13;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "13 -> 14;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "14 -> 15;
    _np_closure[threadIdx.x].src = src;
    // FP: "15 -> 16;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "18 -> 19;
    // FP: "19 -> 20;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "20 -> 21;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "21 -> 22;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "24 -> 25;
    __syncthreads();
    // FP: "25 -> 26;
    while (true)
    {
      // FP: "26 -> 27;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "29 -> 30;
      __syncthreads();
      // FP: "30 -> 31;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "31 -> 32;
        __syncthreads();
        // FP: "32 -> 33;
        break;
      }
      // FP: "34 -> 35;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "37 -> 38;
      __syncthreads();
      // FP: "38 -> 39;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "39 -> 40;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "42 -> 43;
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "43 -> 44;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type jj;
        jj = ns +_np_j;
        {
          index_type dst;
          unsigned int new_dist;
          unsigned int old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
          old_dist = atomicMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            sum_retval.do_return( 1);
          }
        }
      }
      // FP: "56 -> 57;
      __syncthreads();
    }
    // FP: "58 -> 59;

    // FP: "59 -> 60;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "60 -> 61;
      const int _np_laneid = hipcub::LaneId();
      // FP: "61 -> 62;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type jj;
          jj = _np_w_start +_np_ii;
          {
            index_type dst;
            unsigned int new_dist;
            unsigned int old_dist;
            dst = graph.getAbsDestination(jj);
            new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
            old_dist = atomicMin(&p_dist_current[dst], new_dist);
            if (old_dist > new_dist)
            {
              sum_retval.do_return( 1);
            }
          }
        }
      }
      // FP: "84 -> 85;
      __syncthreads();
      // FP: "85 -> 86;
    }

    // FP: "86 -> 87;
    __syncthreads();
    // FP: "87 -> 88;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "88 -> 89;
    while (_np.work())
    {
      // FP: "89 -> 90;
      int _np_i =0;
      // FP: "90 -> 91;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "91 -> 92;
      __syncthreads();
      // FP: "92 -> 93;

      // FP: "93 -> 94;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type jj;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        jj= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          unsigned int new_dist;
          unsigned int old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
          old_dist = atomicMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            sum_retval.do_return( 1);
          }
        }
      }
      // FP: "107 -> 108;
      _np.execute_round_done(ITSIZE);
      // FP: "108 -> 109;
      __syncthreads();
    }
    // FP: "110 -> 111;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "112 -> 113;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, const unsigned int & local_infinity, unsigned int local_src_node, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, local_infinity, local_src_node, ctx->dist_current.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_all_cuda(const unsigned int & local_infinity, unsigned int local_src_node, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->nowned, local_infinity, local_src_node, ctx);
  // FP: "2 -> 3;
}
void SSSP_cuda(unsigned int  __begin, unsigned int  __end, int & __retval, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  *(ctx->p_retval.cpu_wr_ptr()) = __retval;
  // FP: "5 -> 6;
  ctx->sum_retval.rv = ctx->p_retval.gpu_wr_ptr();
  // FP: "6 -> 7;
  SSSP <<<blocks, __tb_SSSP>>>(ctx->gg, ctx->nowned, __begin, __end, ctx->dist_current.gpu_wr_ptr(), ctx->sum_retval);
  // FP: "7 -> 8;
  check_cuda_kernel;
  // FP: "8 -> 9;
  __retval = *(ctx->p_retval.cpu_rd_ptr());
  // FP: "9 -> 10;
}
void SSSP_all_cuda(int & __retval, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  SSSP_cuda(0, ctx->nowned, __retval, ctx);
  // FP: "2 -> 3;
}