#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_COMP_CURRENT;
unsigned int * P_COMP_OLD;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
static const int __tb_ConnectedComp = TB_SIZE;
static const int __tb_FirstItr_ConnectedComp = TB_SIZE;
__global__ void InitializeGraph(CSRGraph graph, unsigned int nowned, unsigned int * p_comp_current, unsigned int * p_comp_old)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < nowned;
    if (pop)
    {
      p_comp_current[src] = graph.node_data[src];
      p_comp_old[src] = graph.node_data[src];
    }
  }
  // FP: "8 -> 9;
}
__global__ void FirstItr_ConnectedComp(CSRGraph graph, unsigned int nowned, unsigned int * p_comp_current, unsigned int * p_comp_old)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_FirstItr_ConnectedComp;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = nowned;
  src_rup = (roundup((nowned), (blockDim.x)));
  for (index_type src = 0 + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < nowned;
    // FP: "7 -> 8;
    if (pop)
    {
      p_comp_old[src] = p_comp_current[src];
    }
    // FP: "10 -> 11;
    // FP: "13 -> 14;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "14 -> 15;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "15 -> 16;
    _np_closure[threadIdx.x].src = src;
    // FP: "16 -> 17;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "19 -> 20;
    // FP: "20 -> 21;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "21 -> 22;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "22 -> 23;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "25 -> 26;
    __syncthreads();
    // FP: "26 -> 27;
    while (true)
    {
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      __syncthreads();
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        __syncthreads();
        break;
      }
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      __syncthreads();
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type jj;
        jj = ns +_np_j;
        {
          index_type dst;
          unsigned int new_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = p_comp_current[src];
          atomicMin(&p_comp_current[dst], new_dist);
        }
      }
      __syncthreads();
      // FP: "54 -> 27;
    }
    // FP: "55 -> 56;

    // FP: "56 -> 57;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "57 -> 58;
      const int _np_laneid = hipcub::LaneId();
      // FP: "58 -> 59;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type jj;
          jj = _np_w_start +_np_ii;
          {
            index_type dst;
            unsigned int new_dist;
            dst = graph.getAbsDestination(jj);
            new_dist = p_comp_current[src];
            atomicMin(&p_comp_current[dst], new_dist);
          }
        }
      }
      // FP: "77 -> 78;
      __syncthreads();
      // FP: "78 -> 79;
    }

    // FP: "79 -> 80;
    __syncthreads();
    // FP: "80 -> 81;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "81 -> 82;
    while (_np.work())
    {
      // FP: "82 -> 83;
      int _np_i =0;
      // FP: "83 -> 84;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "84 -> 85;
      __syncthreads();
      // FP: "85 -> 86;

      // FP: "86 -> 87;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type jj;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        jj= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          unsigned int new_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = p_comp_current[src];
          atomicMin(&p_comp_current[dst], new_dist);
        }
      }
      // FP: "96 -> 97;
      _np.execute_round_done(ITSIZE);
      // FP: "97 -> 98;
      __syncthreads();
      // FP: "98 -> 82;
    }
    // FP: "99 -> 100;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
    // FP: "100 -> 6;
  }
  // FP: "101 -> 102;
}
__global__ void ConnectedComp(CSRGraph graph, unsigned int nowned, unsigned int * p_comp_current, unsigned int * p_comp_old, Any any_retval)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_ConnectedComp;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = nowned;
  src_rup = (roundup((nowned), (blockDim.x)));
  for (index_type src = 0 + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < nowned;
    // FP: "7 -> 8;
    if (pop)
    {
      if (p_comp_old[src] > p_comp_current[src])
      {
        p_comp_old[src] = p_comp_current[src];
        any_retval.return_( 1);
      }
      else
      {
        pop = false;
      }
    }
    // FP: "13 -> 14;
    // FP: "16 -> 17;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "17 -> 18;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "18 -> 19;
    _np_closure[threadIdx.x].src = src;
    // FP: "19 -> 20;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "22 -> 23;
    // FP: "23 -> 24;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "24 -> 25;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "25 -> 26;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "28 -> 29;
    __syncthreads();
    // FP: "29 -> 30;
    while (true)
    {
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      __syncthreads();
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        __syncthreads();
        break;
      }
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      __syncthreads();
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type jj;
        jj = ns +_np_j;
        {
          index_type dst;
          unsigned int new_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = p_comp_current[src];
          atomicMin(&p_comp_current[dst], new_dist);
        }
      }
      __syncthreads();
      // FP: "57 -> 30;
    }
    // FP: "58 -> 59;

    // FP: "59 -> 60;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "60 -> 61;
      const int _np_laneid = hipcub::LaneId();
      // FP: "61 -> 62;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type jj;
          jj = _np_w_start +_np_ii;
          {
            index_type dst;
            unsigned int new_dist;
            dst = graph.getAbsDestination(jj);
            new_dist = p_comp_current[src];
            atomicMin(&p_comp_current[dst], new_dist);
          }
        }
      }
      // FP: "80 -> 81;
      __syncthreads();
      // FP: "81 -> 82;
    }

    // FP: "82 -> 83;
    __syncthreads();
    // FP: "83 -> 84;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "84 -> 85;
    while (_np.work())
    {
      // FP: "85 -> 86;
      int _np_i =0;
      // FP: "86 -> 87;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "87 -> 88;
      __syncthreads();
      // FP: "88 -> 89;

      // FP: "89 -> 90;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type jj;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        jj= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          unsigned int new_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = p_comp_current[src];
          atomicMin(&p_comp_current[dst], new_dist);
        }
      }
      // FP: "99 -> 100;
      _np.execute_round_done(ITSIZE);
      // FP: "100 -> 101;
      __syncthreads();
      // FP: "101 -> 85;
    }
    // FP: "102 -> 103;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
    // FP: "103 -> 6;
  }
  // FP: "105 -> 106;
}
void InitializeGraph_cuda(struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, ctx->comp_current.gpu_wr_ptr(), ctx->comp_old.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void FirstItr_ConnectedComp_cuda(struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  FirstItr_ConnectedComp <<<blocks, __tb_FirstItr_ConnectedComp>>>(ctx->gg, ctx->nowned, ctx->comp_current.gpu_wr_ptr(), ctx->comp_old.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void ConnectedComp_cuda(int & __retval, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(ctx->gg, blocks, threads);
  // FP: "4 -> 5;
  *(ctx->p_retval.cpu_wr_ptr()) = __retval;
  // FP: "5 -> 6;
  ctx->any_retval.rv = ctx->p_retval.gpu_wr_ptr();
  // FP: "6 -> 7;
  ConnectedComp <<<blocks, __tb_ConnectedComp>>>(ctx->gg, ctx->nowned, ctx->comp_current.gpu_wr_ptr(), ctx->comp_old.gpu_wr_ptr(), ctx->any_retval);
  // FP: "7 -> 8;
  check_cuda_kernel;
  // FP: "8 -> 9;
  __retval = *(ctx->p_retval.cpu_rd_ptr());
  // FP: "9 -> 10;
}
