#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
int * P_NOUT;
float * P_VALUE;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
static const int __tb_InitializeGraph = TB_SIZE;
__global__ void ResetGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, int * p_nout, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_value[src] = 0;
      p_nout[src] = 0;
    }
  }
  // FP: "8 -> 9;
}
__global__ void InitializeGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float  local_alpha, int * p_nout, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_InitializeGraph;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
      p_value[src] = local_alpha;
    }
    // FP: "10 -> 11;
    // FP: "13 -> 14;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "14 -> 15;
    __shared__ struct { ; } _np_closure [TB_SIZE];
    // FP: "15 -> 16;
    // FP: "16 -> 17;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "19 -> 20;
    // FP: "20 -> 21;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "21 -> 22;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "22 -> 23;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "25 -> 26;
    __syncthreads();
    // FP: "26 -> 27;
    while (true)
    {
      // FP: "27 -> 28;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "30 -> 31;
      __syncthreads();
      // FP: "31 -> 32;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "32 -> 33;
        __syncthreads();
        // FP: "33 -> 34;
        break;
      }
      // FP: "35 -> 36;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "38 -> 39;
      __syncthreads();
      // FP: "39 -> 40;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "40 -> 41;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "43 -> 44;
      assert(nps.tb.src < __kernel_tb_size);
      // FP: "44 -> 45;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicAdd(&p_nout[dst], 1);
        }
      }
      // FP: "51 -> 52;
      __syncthreads();
    }
    // FP: "53 -> 54;

    // FP: "54 -> 55;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "55 -> 56;
      const int _np_laneid = hipcub::LaneId();
      // FP: "56 -> 57;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            atomicAdd(&p_nout[dst], 1);
          }
        }
      }
      // FP: "73 -> 74;
      __syncthreads();
      // FP: "74 -> 75;
    }

    // FP: "75 -> 76;
    __syncthreads();
    // FP: "76 -> 77;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "77 -> 78;
    while (_np.work())
    {
      // FP: "78 -> 79;
      int _np_i =0;
      // FP: "79 -> 80;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "80 -> 81;
      __syncthreads();
      // FP: "81 -> 82;

      // FP: "82 -> 83;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicAdd(&p_nout[dst], 1);
        }
      }
      // FP: "90 -> 91;
      _np.execute_round_done(ITSIZE);
      // FP: "91 -> 92;
      __syncthreads();
    }
    // FP: "93 -> 94;
    assert(threadIdx.x < __kernel_tb_size);
  }
  // FP: "95 -> 96;
}
__global__ void PageRank(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float  local_alpha, float local_tolerance, int * p_nout, float * p_value, Sum ret_val)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  typedef hipcub::BlockReduce<int, TB_SIZE> _br;
  __shared__ _br::TempStorage _ts;
  ret_val.thread_entry();
  float sum;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    index_type nbr_end;
    float pr_value;
    float diff;
    bool pop  = src < __end;
    if (pop)
    {
      sum = 0;
    }
    if (!pop)
    {
      continue;
    }
    nbr_end = (graph).getFirstEdge((src) + 1);
    for (index_type nbr = (graph).getFirstEdge(src) + 0; nbr < nbr_end; nbr += 1)
    {
      index_type dst;
      unsigned int dnout;
      dst = graph.getAbsDestination(nbr);
      dnout = p_nout[dst];
      if (dnout > 0)
      {
        sum += p_value[dst]/dnout;
      }
    }
    pr_value = sum*(1.0 - local_alpha) + local_alpha;
    diff = fabs(pr_value - p_value[src]);
    if (diff > local_tolerance)
    {
      p_value[src] = pr_value;
      ret_val.do_return( 1);
      continue;
    }
  }
  ret_val.thread_exit<_br>(_ts);
}
void ResetGraph_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  ResetGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, ctx->nout.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void ResetGraph_all_cuda(struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->nowned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, const float & local_alpha, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, __tb_InitializeGraph>>>(ctx->gg, ctx->nowned, __begin, __end, local_alpha, ctx->nout.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_all_cuda(const float & local_alpha, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->nowned, local_alpha, ctx);
  // FP: "2 -> 3;
}
void PageRank_cuda(unsigned int  __begin, unsigned int  __end, int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<int> retval = Shared<int>(1);
  Sum _rv;
  *(retval.cpu_wr_ptr()) = 0;
  _rv.rv = retval.gpu_wr_ptr();
  PageRank <<<blocks, threads>>>(ctx->gg, ctx->nowned, __begin, __end, local_alpha, local_tolerance, ctx->nout.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr(), _rv);
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
  __retval = *(retval.cpu_rd_ptr());
  // FP: "7 -> 8;
}
void PageRank_all_cuda(int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->nowned, __retval, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}