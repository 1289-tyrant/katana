#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=False $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=1 $ instrument=set([]) $ unroll=[] $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=False $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_DIST_CURRENT;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
__global__ void InitializeGraph(CSRGraph graph, int  nowned, unsigned long local_infinity, int local_src_node, unsigned int * p_dist_current)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    p_dist_current[src] = (src == local_src_node) ? 0 : local_infinity;
  }
}
__global__ void BFS(CSRGraph graph, int  nowned, unsigned int * p_dist_current, Any any_retval)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    unsigned int current_min;
    index_type jj_end;
    current_min = p_dist_current[src];
    jj_end = (graph).getFirstEdge((src) + 1);
    for (index_type jj = (graph).getFirstEdge(src) + 0; jj < jj_end; jj += 1)
    {
      index_type dst;
      unsigned int new_dist;
      dst = graph.getAbsDestination(jj);
      new_dist = p_dist_current[dst] + 1;
      if (current_min > new_dist)
      {
        current_min = new_dist;
      }
    }
    if (p_dist_current[src] > current_min)
    {
      p_dist_current[src] = current_min;
      any_retval.return_( 1);
    }
  }
}
void InitializeGraph_cuda(int local_src_node, unsigned long local_infinity, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_infinity, local_src_node, ctx->dist_current.gpu_wr_ptr());
  check_cuda_kernel;
}
void BFS_cuda(int & __retval, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  *(ctx->p_retval.cpu_wr_ptr()) = __retval;
  ctx->any_retval.rv = ctx->p_retval.gpu_wr_ptr();
  BFS <<<blocks, threads>>>(ctx->gg, ctx->nowned, ctx->dist_current.gpu_wr_ptr(), ctx->any_retval);
  check_cuda_kernel;
  __retval = *(ctx->p_retval.cpu_rd_ptr());
}