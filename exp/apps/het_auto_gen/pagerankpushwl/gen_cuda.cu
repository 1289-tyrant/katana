#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=True $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['wp', 'fg']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_NOUT;
float * P_RESIDUAL;
float * P_VALUE;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
__global__ void InitializeGraph(CSRGraph graph, int  nowned, const float  local_alpha, unsigned int * p_nout, float * p_residual, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    p_value[src] = local_alpha;
    p_nout[src] = graph.getOutDegree(src);
    if (p_nout[src] > 0)
    {
      float delta;
      index_type nbr_end;
      delta = p_value[src]*(1-local_alpha)/p_nout[src];
      nbr_end = (graph).getFirstEdge((src) + 1);
      for (index_type nbr = (graph).getFirstEdge(src) + 0; nbr < nbr_end; nbr += 1)
      {
        index_type dst;
        dst = graph.getAbsDestination(nbr);
        atomicAdd(&p_residual[dst], delta);
      }
    }
  }
}
__global__ void PageRank(CSRGraph graph, int  nowned, const float  local_alpha, float local_tolerance, unsigned int * p_nout, float * p_residual, float * p_value, Worklist2 in_wl, Worklist2 out_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  if (tid == 0)
    in_wl.reset_next_slot();

  index_type wlvertex_end;
  wlvertex_end = *((volatile index_type *) (in_wl).dindex);
  for (index_type wlvertex = 0 + tid; wlvertex < wlvertex_end; wlvertex += nthreads)
  {
    int src;
    bool pop;
    float residual_old;
    pop = (in_wl).pop_id(wlvertex, src);
    residual_old = atomicExch(&p_residual[src], 0.0);
    p_value[src] += residual_old;
    if (p_nout[src] > 0)
    {
      float delta;
      index_type nbr_end;
      delta = residual_old*(1-local_alpha)/p_nout[src];
      nbr_end = (graph).getFirstEdge((src) + 1);
      for (index_type nbr = (graph).getFirstEdge(src) + 0; nbr < nbr_end; nbr += 1)
      {
        index_type dst;
        float dst_residual_old;
        dst = graph.getAbsDestination(nbr);
        dst_residual_old = atomicAdd(&p_residual[dst], delta);
        if ((dst_residual_old <= local_tolerance) && ((dst_residual_old + delta) >= local_tolerance))
        {
          index_type _start_46;
          _start_46 = (out_wl).setup_push_warp_one();;
          (out_wl).do_push(_start_46, 0, dst);
        }
      }
    }
  }
}
void InitializeGraph_cuda(const float & local_alpha, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_alpha, ctx->nout.gpu_wr_ptr(), ctx->residual.gpu_wr_ptr(), ctx->value.gpu_wr_ptr());
  check_cuda_kernel;
}
void PageRank_cuda(const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  ctx->in_wl.update_gpu(ctx->shared_wl->num_in_items);
  ctx->out_wl.will_write();
  ctx->out_wl.reset();
  PageRank <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_alpha, local_tolerance, ctx->nout.gpu_wr_ptr(), ctx->residual.gpu_wr_ptr(), ctx->value.gpu_wr_ptr(), ctx->in_wl, ctx->out_wl);
  check_cuda_kernel;
  ctx->out_wl.update_cpu();
  ctx->shared_wl->num_out_items = ctx->out_wl.nitems();
}
