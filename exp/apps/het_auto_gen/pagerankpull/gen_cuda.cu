#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=False $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=1 $ instrument=set([]) $ unroll=[] $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=False $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
int * P_NOUT;
float * P_VALUE;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
__global__ void InitializeGraph(CSRGraph graph, int  nowned, const float  local_alpha, int * p_nout, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    index_type nbr_end;
    p_value[src] = 1.0 - local_alpha;
    nbr_end = (graph).getFirstEdge((src) + 1);
    for (index_type nbr = (graph).getFirstEdge(src) + 0; nbr < nbr_end; nbr += 1)
    {
      index_type dst;
      dst = graph.getAbsDestination(nbr);
      atomicAdd(&p_nout[dst], 1);
    }
  }
}
__global__ void PageRank_pull(CSRGraph graph, int  nowned, const float  local_alpha, const float  local_tolerance, int * p_nout, float * p_value, Any any_retval)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    float sum;
    index_type nbr_end;
    float pr_value;
    float diff;
    sum = 0;
    nbr_end = (graph).getFirstEdge((src) + 1);
    for (index_type nbr = (graph).getFirstEdge(src) + 0; nbr < nbr_end; nbr += 1)
    {
      index_type dst;
      unsigned int dnout;
      dst = graph.getAbsDestination(nbr);
      dnout = p_nout[dst];
      if (dnout > 0)
      {
        sum += p_value[dst]/dnout;
      }
    }
    pr_value = sum*(1.0 - local_alpha) + local_alpha;
    diff = fabs(pr_value - p_value[src]);
    if (diff > local_tolerance)
    {
      p_value[src] = pr_value;
      any_retval.return_( 1);
    }
  }
}
void InitializeGraph_cuda(const float & local_alpha, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_alpha, ctx->nout.gpu_wr_ptr(), ctx->value.gpu_wr_ptr());
  check_cuda_kernel;
}
void PageRank_pull_cuda(int & __retval, const float & local_alpha, const float & local_tolerance, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  *(ctx->p_retval.cpu_wr_ptr()) = __retval;
  ctx->any_retval.rv = ctx->p_retval.gpu_wr_ptr();
  PageRank_pull <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_alpha, local_tolerance, ctx->nout.gpu_wr_ptr(), ctx->value.gpu_wr_ptr(), ctx->any_retval);
  check_cuda_kernel;
  __retval = *(ctx->p_retval.cpu_rd_ptr());
}