#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=False $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=1 $ instrument=set([]) $ unroll=[] $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=False $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
unsigned int * P_COMP_CURRENT;
#include "kernels/reduce.cuh"
#include "gen_cuda.cuh"
__global__ void InitializeGraph(CSRGraph graph, int  nowned, unsigned long local_offset, unsigned int * p_comp_current)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    p_comp_current[src] = src + local_offset;
  }
}
__global__ void ConnectedComp(CSRGraph graph, int  nowned, unsigned int * p_comp_current, Any any_retval)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  src_end = nowned;
  for (index_type src = 0 + tid; src < src_end; src += nthreads)
  {
    unsigned int sdist;
    index_type jj_end;
    sdist = p_comp_current[src];
    jj_end = (graph).getFirstEdge((src) + 1);
    for (index_type jj = (graph).getFirstEdge(src) + 0; jj < jj_end; jj += 1)
    {
      index_type dst;
      unsigned int new_dist;
      unsigned int old_dist;
      dst = graph.getAbsDestination(jj);
      new_dist = sdist;
      old_dist = atomicMin(&p_comp_current[dst], new_dist);
      if (old_dist > new_dist)
      {
        any_retval.return_( 1);
      }
    }
  }
}
void InitializeGraph_cuda(unsigned long local_offset, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  InitializeGraph <<<blocks, threads>>>(ctx->gg, ctx->nowned, local_offset, ctx->comp_current.gpu_wr_ptr());
  check_cuda_kernel;
}
void ConnectedComp_cuda(int & __retval, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  kernel_sizing(ctx->gg, blocks, threads);
  *(ctx->p_retval.cpu_wr_ptr()) = __retval;
  ctx->any_retval.rv = ctx->p_retval.gpu_wr_ptr();
  ConnectedComp <<<blocks, threads>>>(ctx->gg, ctx->nowned, ctx->comp_current.gpu_wr_ptr(), ctx->any_retval);
  check_cuda_kernel;
  __retval = *(ctx->p_retval.cpu_rd_ptr());
}